#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void moveleftKenel(float* vert)
{
    int idx = threadIdx.x;
    vert[idx*3] -= 0.02;
}


bool moveleft(float* vert)
{

    hipError_t cudaStatus = hipSuccess;
    // execute the kernel
    dim3 block(3, 1, 1);
    dim3 grid(1, 1, 1);
    moveleftKenel << < grid, block >> > (vert);


    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    return true;
Error:
    return false;
}

