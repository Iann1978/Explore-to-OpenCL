#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void mxaddKenel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	c[i + j * blockDim.x] = a[i + j * blockDim.x] + b[i + j * blockDim.x];
}

// a(m*q) * b(q*n) = c(m*n)
__global__ void mxmulKenel(int* c, const int* a, const int* b, int q) {


	int m = blockDim.y;
	int n = blockDim.x;
	int i = threadIdx.x;
	int j = threadIdx.y;
	int sum = 0;
	for (int k = 0; k < q; k++) {
		sum += a[k + j * q] * b[i + k * n];
	}
	c[i + j * n] = sum;
}
// a(m*q) * b(q*n) = c(m*n)
hipError_t mxmul(int* c, const int* a, const int* b, int m, int q, int n) {

	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	cudaStatus = hipMalloc((void**)&dev_c, m * n * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_a, m * q * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_b, q * n * sizeof(int));

	cudaStatus = hipMemcpy(dev_a, a, m * q * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_b, b, q * n * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimBlock(n, m);
	mxmulKenel << <1, dimBlock >> >(dev_c, dev_a, dev_b, q);

	cudaStatus = hipGetLastError();
	cudaStatus = hipDeviceSynchronize();

	cudaStatus = hipMemcpy(c, dev_c, m * n * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

hipError_t mxadd(int* c, const int* a, const int* b, int w, int h) {

	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	cudaStatus = hipMalloc((void**)&dev_c, w * h * sizeof(int));	
	cudaStatus = hipMalloc((void**)&dev_a, w * h * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_b, w * h * sizeof(int));

	cudaStatus = hipMemcpy(dev_a, a, w * h * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_b, b, w * h * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimBlock(w, h);
	mxaddKenel << <1, dimBlock >> >(dev_c, dev_a, dev_b);

	cudaStatus = hipGetLastError();
	cudaStatus = hipDeviceSynchronize();

	cudaStatus = hipMemcpy(c, dev_c, w * h * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}