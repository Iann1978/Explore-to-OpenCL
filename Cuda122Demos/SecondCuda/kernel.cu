#include "hip/hip_runtime.h"
﻿// https://zhuanlan.zhihu.com/p/34587739
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


void GetCudaInfo() {
	int dev = 0;
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, dev);
	printf("Using Device %d: %s\n", dev, devProp.name);
	printf("Compute capability: %d.%d\n", devProp.major, devProp.minor);
	printf("Clock rate: %d\n", devProp.clockRate);
	printf("SM count: %d\n", devProp.multiProcessorCount);
	printf("Shared Memory: %d\n", devProp.sharedMemPerBlock);
	printf("Max Threads per Block: %d\n", devProp.maxThreadsPerBlock);
	printf("Max Threads per MultiProcessor: %d\n", devProp.maxThreadsPerMultiProcessor);

	printf("Device copy overlap: ");
	if (devProp.deviceOverlap)
		printf("Enabled\n");
	else
		printf("Disabled\n");


}

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKenel(int* c, const int* a, const int* b) {
	int idx = threadIdx.x;
	c[idx] = a[idx] + b[idx];
}


hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size) {
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice(0) error");
		goto Error;
	}

	cudaStatus = hipMalloc(&dev_a, sizeof(int) * size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_a error");
		goto Error;
	}

	cudaStatus = hipMalloc(&dev_b, sizeof(int) * size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_b error");
		goto Error;
	}

	cudaStatus = hipMalloc(&dev_c, sizeof(int) * size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_c error");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, sizeof(int) * size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy dev_a error");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, sizeof(int) * size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy dev_b error");
		goto Error;
	}

	addKenel<<<1, size>>>(dev_c, dev_a, dev_b);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKenel error");
		goto Error;
	}

	cudaStatus = hipMemcpy(c, dev_c, sizeof(int) * size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy c error");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return cudaStatus;
}


int main() {
	GetCudaInfo();

	int a[] = { 1,2,3,4,5 };
	int b[] = { 2,3,4,5,6 };
	int c[] = { 0,0,0,0,0 };

	hipError_t cudaStatus = addWithCuda(c, a, b, 5);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	printf("{1,2,3,4,5} + {2,3,4,5,6} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return;
}

