#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void moveLeft(float* vert)
{
    int idx = threadIdx.y * blockDim.x + threadIdx.x;
    vert[idx] = vert[idx] + 0.0051;
}

bool launch_kernel(float* pos, int width, int height)
{

    hipError_t cudaStatus = hipSuccess;
    // execute the kernel
    dim3 grid(1, 1, 1);
    dim3 block(width, height, 1); 
    moveLeft << < grid, block >> > (pos);


    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    return true;
Error:
    return false;
}
