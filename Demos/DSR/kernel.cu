#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void moveLeft(float* vert)
{
    vert[0] = 0.2;
}

bool launch_kernel(float* pos)
{

    hipError_t cudaStatus = hipSuccess;
    // execute the kernel
    dim3 block(1, 1, 1);
    dim3 grid(1, 1, 1);
    moveLeft << < grid, block >> > (pos);


    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    return true;
Error:
    return false;
}
