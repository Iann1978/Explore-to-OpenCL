#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>



__global__ void Add0001(float* vert, int width, int height)
{
    if (blockIdx.x >= width)
        return;
    if (blockIdx.y >= height)
        return;

    int idx = blockIdx.y * gridDim.x + blockIdx.x;
    //if (idx < 100)
    {
        vert[idx] = vert[idx] + 0.001 * blockIdx.x;
    }

}

bool launch_kernel(float* pos, int width, int height)
{

    int blockWidth = width < 32 ? width : 32;
    int blockHeight = height < 32 ? height : 32;
    int gridWidth = width / blockWidth;
    int gridHeight = height / blockHeight;


    hipError_t cudaStatus = hipSuccess;
    // execute the kernel
    dim3 grid(width, height, 1);
    dim3 block(1, 1, 1);
    Add0001 << < grid, block >> > (pos, width, height);


    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    return true;
Error:
    return false;
}
