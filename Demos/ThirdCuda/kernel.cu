#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>



#include <iostream>

__global__ void simulateKernel(float* dst, float* src)
{
    int row = threadIdx.x;
    int col = threadIdx.y;
    
    float value0 = src[row * 10 + col];
    float dh = 0;
    for (int drow = -1; drow <= 1; drow++)
    {
        for (int dcol = -1; dcol <= 1; dcol++)
        {
            int index = (row + drow) * 10 + col + dcol;
            if (index < 0)
                continue;
            if (index >= 10 * 10)
                continue;

            float value1 = src[index];

            if (value1 > value0) dh +=1;
            if (value1 < value0) dh -=1;

        }
    }
    dst[row*10+col] = src[row*10+col] + dh;
}

hipError_t simulate(float src[10][10], float dst[10][10])
{
    void* dev_src = 0;
    void* dev_dst = 0;
    hipError_t cudaStatus{};

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc(&dev_src, 10 * 10 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc(&dev_dst, 10 * 10 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_src, src, 10 * 10 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    dim3 block(10, 10);
    simulateKernel<<<1, block>>> ((float*)dev_dst, (float*)dev_src);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "simulateKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching simulateKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(dst , dev_dst, 10 * 10 * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    return cudaStatus;
Error:
    hipFree(dev_src);
    hipFree(dev_dst);
    return cudaStatus;
}


void flowtest()
{
    float src[10][10];
    for (int row = 0; row < 10; row++)
    {
        for (int col = 0; col < 10; col++)
        {
            src[row][col] = 5;
        }
    }

    float dst[10][10];
    for (int row = 0; row < 10; row++)
    {
        for (int col = 0; col < 10; col++)
        {
            dst[row][col] = 0;
        }
    }
    src[5][5] = 10;

    std::cout << "before simulate " << std::endl;
    std::cout << "src: " << std::endl;
    for (int row = 0; row < 10; row++)
    {
        for (int col = 0; col < 10; col++)
        {
            std::cout << src[row][col] << ",";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;

    std::cout << "dst: " << std::endl;
    for (int row = 0; row < 10; row++)
    {
        for (int col = 0; col < 10; col++)
        {
            std::cout << dst[row][col] << ",";
        }
        std::cout << std::endl;
    }

    simulate(src, dst);
    std::cout << std::endl;
    std::cout << std::endl;


    std::cout << "after simulate " << std::endl;
    std::cout << "src: " << std::endl;
    for (int row = 0; row < 10; row++)
    {
        for (int col = 0; col < 10; col++)
        {
            std::cout << src[row][col] << ",";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;

    std::cout << "dst: " << std::endl;
    for (int row = 0; row < 10; row++)
    {
        for (int col = 0; col < 10; col++)
        {
            std::cout << dst[row][col] << ",";
        }
        std::cout << std::endl;
    }

}
int main()
{

    flowtest();

  

    return 0;
}

